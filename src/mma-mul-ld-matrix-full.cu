#include <cassert>
#include <cmath>
#include <cstdio>
#include <hip/hip_fp16.h>
// A is 16 x 16
// B is 16 x 8
// C is 16 x 8

#include <hip/hip_runtime.h>

template <typename T> struct Afrag_16x16 {
  static constexpr size_t ne = 8;

  T x[ne];

  static __device__ size_t get_row(int tid, int l) {
    int group_id = tid >> 2; // same as /4

    return group_id + 8 * ((l / 2) % 2);
  }

  static __device__ size_t get_col(int tid, int l) {
    return 2 * (tid % 4) + (l % 2) + 8 * (l / 4);
  }
};

// col major?
template <typename T> struct Bfrag_16x8 {
  static constexpr size_t ne = 4;
  T x[ne] = {};
  static __device__ size_t get_row(int tid, int l) {
    return (tid % 4) * 2 + (l % 2) + 8 * (l / 2);
  }

  static __device__ size_t get_col(int tid, int l) { return tid >> 2; }
};

template <typename T> struct CFrag_16x8 {
  static constexpr size_t ne = 4;
  T x[ne] = {};

  static __device__ size_t get_row(int tid, int l) {
    return (tid >> 2) + 8 * (l / 2);
  }

  static __device__ size_t get_col(int tid, int l) {
    assert(l < ne);
    return 2 * (tid % 4) + (l % 2);
  }
};

__global__ void mmaKernel(const half *A, const half *B, float *C, int M, int N,
                          int K) {
  Afrag_16x16<half> a_tile;
  Bfrag_16x8<half> b_tile;
  CFrag_16x8<float> c_tile;

  const int tid = threadIdx.x;

  __shared__ alignas(16) half A_shared[16][16];
  __shared__ alignas(16) half B_shared[16][8];
  __shared__ alignas(16) half C_shared[16][8];

  const int lane = tid & 31;

  int c_row = blockIdx.y * 16;
  int c_col = blockIdx.x * 8;

  A += c_row * K;
  B += c_col;


  for(int k_idx = 0; k_idx < K; k_idx += 16) {

    if (lane < 16) {
      int row = lane;

      for(int idx = 0; idx < 8; ++idx) {
          A_shared[row][idx] = A[row*K + idx];
      }

      for(int idx = 0; idx < 8; ++idx) {
          A_shared[row][idx + 8] = A[row*K + 8 + idx];
      }

      for(int idx = 0 ; idx < 8; ++idx) {
          B_shared[row][idx] = B[row*N + idx];
      }
    }

    //next iteration move down
    A += 16;
    B += 16*N;

    int *a_regs = (int *)a_tile.x;
    int *b_regs = (int *)b_tile.x;

    int lane_id = tid;
    uint32_t a_addr = __cvta_generic_to_shared(
        &A_shared[(lane_id % 16)][(lane_id / 16) * 8]);
    uint32_t b_addr = __cvta_generic_to_shared(
        &B_shared[(lane_id % 16)]);

    asm volatile("ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
                "{%0, %1, %2, %3}, [%4];"
                : "=r"(a_regs[0]), "=r"(a_regs[1]), "=r"(a_regs[2]),
                  "=r"(a_regs[3])
                : "r"(a_addr));

    asm volatile("ldmatrix.sync.aligned.m8n8.x2.shared.trans.b16 "
                "{%0, %1}, [%2];"
                : "=r"(b_regs[0]), "=r"(b_regs[1])
                : "r"(b_addr));

    asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
                "{%0, %1, %2, %3}, "
                "{%4, %5, %6, %7}, "
                "{%8, %9}, "
                "{%0, %1, %2, %3};\n"
                : "+f"(c_tile.x[0]), "+f"(c_tile.x[1]), "+f"(c_tile.x[2]),
                  "+f"(c_tile.x[3])
                : "r"(a_regs[0]), "r"(a_regs[1]), "r"(a_regs[2]), "r"(a_regs[3]),
                  "r"(b_regs[0]), "r"(b_regs[1]));
  }    
  
  for (int i = 0; i < c_tile.ne; ++i) {
      int row = c_row + c_tile.get_row(tid, i);
      int col = c_col + c_tile.get_col(tid, i);

      C[row * N + col] += c_tile.x[i];
   }
}

__global__ void naiveKernel(const half *a, const half *b, float *c, int M,
                            int N, int K) {

  int row = blockIdx.y;
  int col = blockIdx.x;

  float tmp = 0;

  for (int i = 0; i < K; ++i) {
    tmp += (float)(a[row * K + i] * b[i * N + col]);
  }

  c[row * N + col] = tmp;
}

int main() {

  half *a;
  half *b;
  float *c;
  float *d;

  const int M = 1024;
  const int N = 1024;
  const int K = 512;

  hipMallocManaged(&a, M * K * sizeof(half));
  hipMallocManaged(&b, K * N * sizeof(half));
  hipMallocManaged(&c, M * N * sizeof(float));
  hipMallocManaged(&d, M * N * sizeof(float));

  for (int i = 0; i < M * K; ++i) {
    a[i] = __float2half((float)rand() / RAND_MAX);
  }

  for (int i = 0; i < K * N; ++i) {
    b[i] = __float2half((float)rand() / RAND_MAX);
  }

  dim3 grid(N, M);
  dim3 block(1, 1, 1);

  naiveKernel<<<grid, block>>>(a, b, c, M, N, K);

  dim3 mma_grid(N/8, M/16);
  dim3 mma_block(32, 1, 1);
  mmaKernel<<<mma_grid, mma_block>>>(a, b, d, M, N, K);

  hipDeviceSynchronize();

  float rmse = 0.;
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float err = std::abs(d[i * N + j] - c[i * N + j]);
      rmse += err * err;
    }
  }

  printf("RMSE: %.2f", sqrt(rmse));
}
